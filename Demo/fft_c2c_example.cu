// fft_c2c_example.cu
#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>
#include <cmath>
#include <cstdio>
#include <vector>

#define CHECK_CUDA(call)                                                   \
  do {                                                                     \
    hipError_t err = (call);                                              \
    if (err != hipSuccess) {                                              \
      fprintf(stderr, "CUDA error %s at %s:%d\n", hipGetErrorString(err), \
              __FILE__, __LINE__);                                         \
      exit(EXIT_FAILURE);                                                  \
    }                                                                      \
  } while (0)

#define CHECK_CUFFT(call)                                                    \
  do {                                                                       \
    hipfftResult res = (call);                                                \
    if (res != HIPFFT_SUCCESS) {                                              \
      fprintf(stderr, "cuFFT error %d at %s:%d\n", res, __FILE__, __LINE__); \
      exit(EXIT_FAILURE);                                                    \
    }                                                                        \
  } while (0)

int main() {
  const int N = 8;

  // 准备主机端输入：简单的复数序列（实部为索引，虚部为0）
  std::vector<hipfftComplex> h_in(N);
  for (int i = 0; i < N; ++i) {
    h_in[i].x = static_cast<float>(i);  // real
    h_in[i].y = 0.0f;                   // imag
  }

  // 设备端内存
  hipfftComplex* d_data = nullptr;
  CHECK_CUDA(hipMalloc(&d_data, sizeof(hipfftComplex) * N));
  CHECK_CUDA(hipMemcpy(d_data, h_in.data(), sizeof(hipfftComplex) * N,
                        hipMemcpyHostToDevice));

  // 创建 1D C2C FFT 计划
  hipfftHandle plan;
  CHECK_CUFFT(hipfftPlan1d(&plan, N, HIPFFT_C2C, 1));

  // 前向FFT（in-place）
  CHECK_CUFFT(hipfftExecC2C(plan, d_data, d_data, HIPFFT_FORWARD));

  // 将频域数据拷回主机并打印
  std::vector<hipfftComplex> h_freq(N);
  CHECK_CUDA(hipMemcpy(h_freq.data(), d_data, sizeof(hipfftComplex) * N,
                        hipMemcpyDeviceToHost));
  printf("Forward FFT result:\n");
  for (int i = 0; i < N; ++i) {
    printf("k=%d: (%f, %f)\n", i, h_freq[i].x, h_freq[i].y);
  }

  // 逆向FFT（in-place）
  CHECK_CUFFT(hipfftExecC2C(plan, d_data, d_data, HIPFFT_BACKWARD));

  // 拷回并归一化（cuFFT 的逆变换未归一化，需要除以 N）
  std::vector<hipfftComplex> h_back(N);
  CHECK_CUDA(hipMemcpy(h_back.data(), d_data, sizeof(hipfftComplex) * N,
                        hipMemcpyDeviceToHost));
  for (int i = 0; i < N; ++i) {
    h_back[i].x /= N;
    h_back[i].y /= N;
  }

  printf("\nInverse FFT (normalized) result:\n");
  for (int i = 0; i < N; ++i) {
    printf("n=%d: (%f, %f)\n", i, h_back[i].x, h_back[i].y);
  }

  // 清理
  CHECK_CUFFT(hipfftDestroy(plan));
  CHECK_CUDA(hipFree(d_data));

  return 0;
}

// 编译命令：nvcc -std=c++17 fft_c2c_example.cu -lcufft -o fft_c2c_example

// ❯ nvcc -std=c++17 fft_c2c_example.cu -lcufft -o fft_c2c_example
// ❯ ./fft_c2c_example
// Forward FFT result:
// k=0: (28.000000, 0.000000)
// k=1: (-4.000000, 9.656855)
// k=2: (-4.000000, 4.000000)
// k=3: (-4.000000, 1.656854)
// k=4: (-4.000000, 0.000000)
// k=5: (-4.000000, -1.656854)
// k=6: (-4.000000, -4.000000)
// k=7: (-4.000000, -9.656854)

// Inverse FFT (normalized) result:
// n=0: (0.000000, 0.000000)
// n=1: (1.000000, 0.000000)
// n=2: (2.000000, 0.000000)
// n=3: (3.000000, -0.000000)
// n=4: (4.000000, 0.000000)
// n=5: (5.000000, -0.000000)
// n=6: (6.000000, -0.000000)
// n=7: (7.000000, 0.000000)