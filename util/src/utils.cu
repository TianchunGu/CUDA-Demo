#include <util/utils.cuh>

namespace util {
void __cudaCheck(hipError_t err, const char* file, const int line) {
  if (err != hipSuccess) {
    printf("ERROR: %s:%d, ", file, line);
    printf("CODE:%s, DETAIL:%s\n", hipGetErrorName(err),
           hipGetErrorString(err));
    exit(1);
  }
}

void __kernelCheck(const char* file, const int line) {
  hipError_t err = hipPeekAtLastError();
  if (err != hipSuccess) {
    printf("ERROR: %s:%d, ", file, line);
    printf("CODE:%s, DETAIL:%s\n", hipGetErrorName(err),
           hipGetErrorString(err));
    exit(1);
  }
}

void __cufftCheck(const char* file, const int line) {
  hipError_t err = hipPeekAtLastError();
  if (err != hipSuccess) {
    printf("ERROR: %s:%d, ", file, line);
    printf("CODE:%s, DETAIL:%s\n", hipGetErrorName(err),
           hipGetErrorString(err));
    exit(1);
  }
}

}  // namespace util