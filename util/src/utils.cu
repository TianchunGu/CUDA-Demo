#include <util/utils.cuh>

namespace util {
void __cudaCheck(hipError_t err, const char* file, const int line) {
  if (err != hipSuccess) {
    printf("ERROR: %s:%d, ", file, line);
    printf("CODE:%s, DETAIL:%s\n", hipGetErrorName(err),
           hipGetErrorString(err));
    exit(1);
  }
}

void __kernelCheck(const char* file, const int line) {
  hipError_t err = hipPeekAtLastError();
  if (err != hipSuccess) {
    printf("ERROR: %s:%d, ", file, line);
    printf("CODE:%s, DETAIL:%s\n", hipGetErrorName(err),
           hipGetErrorString(err));
    exit(1);
  }
}

void __cufftCHECK(hipfftResult err, const char* file, const int line) {
  if (err != HIPFFT_SUCCESS) {
    printf("ERROR: %s:%d, ", file, line);
    printf("CODE:%d, DETAIL:%s\n", err, cufftGetErrorString(err));
    exit(1);
  }
}

const char* cufftGetErrorString(hipfftResult err) {
  switch (err) {
    case HIPFFT_SUCCESS:
      return "The cuFFT operation was successful";
    case HIPFFT_INVALID_PLAN:
      return "cuFFT was passed an invalid plan handle";
    case HIPFFT_ALLOC_FAILED:
      return "cuFFT failed to allocate GPU or CPU memory";
    case HIPFFT_INVALID_TYPE:
      return "The cuFFT type provided is unsupported";
    case HIPFFT_INVALID_VALUE:
      return "User specified an invalid pointer or parameter";
    case HIPFFT_INTERNAL_ERROR:
      return "Driver or internal cuFFT library error";
    case HIPFFT_EXEC_FAILED:
      return "Failed to execute an FFT on the GPU";
    case HIPFFT_SETUP_FAILED:
      return "The cuFFT library failed to initialize";
    case HIPFFT_INVALID_SIZE:
      return "User specified an invalid transform size";
    case HIPFFT_UNALIGNED_DATA:
      return "Not currently in use";
    case HIPFFT_INVALID_DEVICE:
      return "Execution of a plan was on different GPU than plan creation";
    case HIPFFT_NO_WORKSPACE:
      return "No workspace has been provided prior to plan execution";
    case HIPFFT_NOT_IMPLEMENTED:
      return "Function does not implement functionality for given parameters";
    case HIPFFT_NOT_SUPPORTED:
      return "Operation is not supported for parameters given";
    default:
      return "Unknown error code";
  }
}

}  // namespace util