#include <util/utils.cuh>

namespace util {
static void __cudaCheck(hipError_t err, const char* file, const int line) {
  if (err != hipSuccess) {
    printf("ERROR: %s:%d, ", file, line);
    printf("CODE:%s, DETAIL:%s\n", hipGetErrorName(err),
           hipGetErrorString(err));
    exit(1);
  }
}

static void __kernelCheck(const char* file, const int line) {
  hipError_t err = hipPeekAtLastError();
  if (err != hipSuccess) {
    printf("ERROR: %s:%d, ", file, line);
    printf("CODE:%s, DETAIL:%s\n", hipGetErrorName(err),
           hipGetErrorString(err));
    exit(1);
  }
}
}  // namespace util