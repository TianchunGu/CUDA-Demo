#include "hip/hip_runtime.h"
#include <cuda_task/cuda_task.cuh>

// 宏定义变换的尺寸和批次数
#define NX 256
#define BATCH 10

__constant__ float c_data;
__constant__ float c_data2 = 6.6f;

namespace cuda_task {
__global__ void kernel_1(void) {
  printf("Constant data c_data = %.2f.\n", c_data);
}

__global__ void kernel_2(int N) {
  int idx = threadIdx.x;
  if (idx < N) {
  }
}

void run_vector_add() {
  int devID = 0;
  hipDeviceProp_t deviceProps;
  util::CUDA_CHECK(hipGetDeviceProperties(&deviceProps, devID));
  std::cout << "运行GPU设备:" << deviceProps.name << std::endl;

  float h_data = 8.8f;
  util::CUDA_CHECK(hipMemcpyToSymbol(HIP_SYMBOL(c_data), &h_data, sizeof(float)));

  dim3 block(1);
  dim3 grid(1);
  kernel_1<<<grid, block>>>();
  util::CUDA_CHECK(hipDeviceSynchronize());
  util::CUDA_CHECK(hipMemcpyFromSymbol(&h_data, HIP_SYMBOL(c_data2), sizeof(float)));
  printf("Constant data h_data = %.2f.\n", h_data);

  util::CUDA_CHECK(hipDeviceReset());
}

/**
 * @brief 执行一次完整的cuFFT变换流程
 *
 * 该函数封装了内存分配、创建计划、执行变换和资源清理的完整三步流程。
 * [cite_start]这个函数的设计基于文档中的示例代码 [cite: 76, 77, 78, 81, 82, 83,
 * 84, 85, 86]。
 */
void perform_cufft_transform() {
  hipfftHandle plan;
  hipfftComplex* data;

  std::cout << "Allocating GPU memory for " << BATCH << " transforms of size "
            << NX << "..." << std::endl;
  util::CUDA_CHECK(
      hipMalloc((void**)&data, sizeof(hipfftComplex) * NX * BATCH));

  std::cout << "Creating cuFFT plan..." << std::endl;
  util::CUFFT_CHECK(hipfftPlan1d(&plan, NX, HIPFFT_C2C, BATCH));

  std::cout << "Executing cuFFT plan..." << std::endl;
  util::CUFFT_CHECK(hipfftExecC2C(plan, data, data, HIPFFT_FORWARD));

  std::cout << "Waiting for GPU to finish..." << std::endl;
  util::CUDA_CHECK(hipDeviceSynchronize());

  std::cout << "Destroying cuFFT plan and freeing memory..." << std::endl;
  util::CUFFT_CHECK(hipfftDestroy(plan));
  util::CUDA_CHECK(hipFree(data));  // 释放GPU内存
}

}  // namespace cuda_task